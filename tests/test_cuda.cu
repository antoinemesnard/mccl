// config
#include <mccl/config/config.hpp>

// core header files
#include <mccl/core/matrix.hpp>
#include <mccl/core/matrix_m4ri.hpp>

// algorithm header files
#include <mccl/algorithm/decoding.hpp>

// tools header files
//#include <mccl/tools/parser.hpp>

// contrib header files
#include <mccl/contrib/string_algo.hpp>
#include <mccl/contrib/thread_pool.hpp>
#include <mccl/contrib/parallel_algorithms.hpp>
#include <mccl/contrib/json.hpp>
#include <mccl/contrib/program_options.hpp>

#include <iostream>
#include <vector>
#include <set>
#include <utility>

#include <hip/hip_runtime.h>

#include "test_utils.hpp"

int main(int, char**)
{
    int status = 0;
    int deviceCount = 0;
    if (hipSuccess != hipGetDeviceCount(&deviceCount))
    {
	LOG_CERR("CUDA: failed hipGetDeviceCount");
	status = 1;
    }
    if (deviceCount == 0)
        LOG_CERR("There is no device supporting CUDA");
    int dev;
    for (dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t deviceProp;
        if (hipSuccess != hipGetDeviceProperties(&deviceProp, dev))
	{
		LOG_CERR("CUDA: failed hipGetDeviceProperties");
		status = 1;
		continue;
	}
        if (dev == 0) {
            if (deviceProp.major == 9999 && deviceProp.minor == 9999)
            {
                LOG_CERR("There is no device supporting CUDA.");
            }
            else if (deviceCount == 1)
            {
                LOG_CERR("There is 1 device supporting CUDA.");
            }
            else
            {
                LOG_CERR("There are " << deviceCount << " devices supporting CUDA");
            }
        }
        printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);
        printf("  Major revision number:                         %d\n",
               deviceProp.major);
        printf("  Minor revision number:                         %d\n",
               deviceProp.minor);
        printf("  Total amount of global memory:                 %u bytes\n",
               deviceProp.totalGlobalMem);
#if CUDART_VERSION >= 2000
        printf("  Number of multiprocessors:                     %d\n",
               deviceProp.multiProcessorCount);
        printf("  Number of cores:                               %d\n",
               8 * deviceProp.multiProcessorCount);
#endif
        printf("  Total amount of constant memory:               %u bytes\n",
               deviceProp.totalConstMem);
        printf("  Total amount of shared memory per block:       %u bytes\n",
               deviceProp.sharedMemPerBlock);
        printf("  Total number of registers available per block: %d\n",
               deviceProp.regsPerBlock);
        printf("  Warp size:                                     %d\n",
               deviceProp.warpSize);
        printf("  Maximum number of threads per block:           %d\n",
               deviceProp.maxThreadsPerBlock);
        printf("  Maximum sizes of each dimension of a block:    %d x %d x %d\n",
               deviceProp.maxThreadsDim[0],
               deviceProp.maxThreadsDim[1],
               deviceProp.maxThreadsDim[2]);
        printf("  Maximum sizes of each dimension of a grid:     %d x %d x %d\n",
               deviceProp.maxGridSize[0],
               deviceProp.maxGridSize[1],
               deviceProp.maxGridSize[2]);
        printf("  Maximum memory pitch:                          %u bytes\n",
               deviceProp.memPitch);
        printf("  Texture alignment:                             %u bytes\n",
               deviceProp.textureAlignment);
        printf("  Clock rate:                                    %.2f GHz\n",
               deviceProp.clockRate * 1e-6f);
#if CUDART_VERSION >= 2000
        printf("  Concurrent copy and execution:                 %s\n",
               deviceProp.deviceOverlap ? "Yes" : "No");
#endif
    }
    if (status)
    {
	LOG_CERR("Tests failed.");
    }
    else
    {
	LOG_CERR("All tests passed.");
    }
    return status;
}
